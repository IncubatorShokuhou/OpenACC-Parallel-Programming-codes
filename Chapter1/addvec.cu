
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 64

__global__ void add( int *a_d, int *b_d, int *c_d ) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) c_d[tid] = a_d[tid] + b_d[tid];
}
int main() 
{
    int a[N], b[N], c[N];
    int *a_d, *b_d, *c_d;
    hipMalloc((void**)&a_d, N * sizeof(int));
    hipMalloc((void**)&b_d, N * sizeof(int));
    hipMalloc((void**)&c_d, N * sizeof(int));
    for(int i = 0; i < N; i++)
    {
        a[i] = 1;
        b[i] = 2;
    }
    hipMemcpy(a_d, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, N*sizeof(int), hipMemcpyHostToDevice);
    dim3 block(32,1,1), grid;
    grid.x = (N+block.x-1)/block.x;
    add<<<grid, block>>>(a_d, b_d, c_d);
    hipMemcpy(c,c_d,N*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++)
        printf("%2d +%2d =%2d\n",a[i],b[i],c[i]);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
return 0;
}